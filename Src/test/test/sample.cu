#include <stdio.h>
#include <time.h>
#include <math.h>
#include <float.h>

#include "hip/hip_runtime.h"


 
const int blocksize = 800; 
const int N = 16; 


const int PROFILE_SIZE = 8376;
const int PROFILE_ARRAY_SIZE = PROFILE_SIZE * 6;
__device__ const int GPU_PROFILE_SIZE = 8376;

const int FRAME_WIDTH = 1024;
const int FRAME_HEIGHT = 768;
const int PART_FRAME_WIDTH = FRAME_WIDTH/10;
const int PART_FRAME_HEIGHT = FRAME_HEIGHT/10;
const int FRAME_DIMENSIONS = FRAME_WIDTH * FRAME_HEIGHT;
const int PARTITIONED_FRAME_DIMENSIONS = (FRAME_WIDTH/10) * (FRAME_HEIGHT/10);
const int FRAME_ARRAY_SIZE = FRAME_DIMENSIONS * 3;
__device__ const int GPU_FRAME_DIMENSIONS = FRAME_DIMENSIONS;

__device__ double X,Y,Z,L,A,B;
__device__ struct xyz
{
	 double X;
	 double Y;
	 double Z;
};

__device__ struct LAB
{
	 double L;
	 double A;
	 double B;
};
__device__ double distance(double L1,double A1,double B1,double L2,double A2,double B2)
{
	 double l, a, b, result,sqresult;
      l = L1 - L2;
      a = A1 - A2;
      b = B1 - B2;
      l = l * l;
      a = a * a;
      b = b * b;
	  result = l + a + b;

	  sqresult = sqrt(result);
	  return sqresult;
}

__device__ void addXYZ(double X1,double Y1,double Z1,double X2,double Y2,double Z2)
{
	X=X1+X2;
	Y=Y1+Y2;
	Z=Z1+Z2;
}
__device__  struct xyz addXYZ_st(double X1,double Y1,double Z1,double X2,double Y2,double Z2)
{
	 struct xyz XYZ;
	XYZ.X=X1+X2;
	XYZ.Y=Y1+Y2;
	XYZ.Z=Z1+Z2;
	return XYZ;
/*
	if(X>0.9504)
		X=0.9504;
	else if (X<0)
		X=0;
	else
		X=X;
	
	if(X>1)
		Y=1
	else if (Y<0)
		Y=0;
	else
		Y=Y;

	if(Z>1.0888)
		Z=1.0888;
	else if (Z<0)
		Z=0;
	else
		Z=Z;
*/
}

// rgb to xyz
//__device__ void RGBToXYZ(unsigned char R,unsigned char G,unsigned char B )
__device__ void RGBToXYZ(int R,int G,int B )
    {
      // by the formula given the the web page http://www.brucelindbloom.com/index.html [XYZ]=[M][RGB]
      //In order to properly use this matrix, the RGB values must be linear and in the nominal range [0.0, 1.0].
      // RGB values may first need conversion (for example, dividing by 255 and then raising them to a power)
      // Where M for D65:	 0.4124564  0.3575761  0.1804375
      //0.2126729  0.7151522  0.0721750
      //0.0193339  0.1191920  0.9503041

      //// to make rgb values linear red, green, blue values
      double rLinear = (double)R / 255.0;
      double gLinear = (double)G / 255.0;
      double bLinear = (double)B / 255.0;

      // convert to a sRGB form

      //double r =  Math.pow((rLinear ), 2.2) ;
      //double g =  Math.pow((gLinear ), 2.2) ;
      //double b = Math.pow((bLinear ), 2.2) ;
      double r, g, b;

      if (rLinear > 0.04045)
        r = pow(((rLinear + 0.055) / 1.055), 2.2);
      else
        r = rLinear / 12.92;

      if (gLinear > 0.04045)
        g = pow(((gLinear + 0.055) / 1.055), 2.2);
      else
        g = gLinear / 12.92;

      if (bLinear > 0.04045)
        b = pow(((bLinear + 0.055) / 1.055), 2.2);
      else
        b = bLinear / 12.92;


		X=(r * 0.4124564 + g * 0.3575761 + b * 0.1804375);
        Y=(r * 0.2126729 + g * 0.7151522 + b * 0.0721750);
        Z=(r * 0.0193339 + g * 0.1191920 + b * 0.9503041);

		/*
	if(X>0.9504)
		X=0.9504;
	else if (X<0)
		X=0;
	else
		X=X;
	
	if(X>1)
		Y=1
	else if (Y<0)
		Y=0;
	else
		Y=Y;

	if(Z>1.0888)
		Z=1.0888;
	else if (Z<0)
		Z=0;
	else
		Z=Z;
*/
    }
__device__ struct xyz RGBToXYZ_St(int R,int G,int B )
    {
      // by the formula given the the web page http://www.brucelindbloom.com/index.html [XYZ]=[M][RGB]
      //In order to properly use this matrix, the RGB values must be linear and in the nominal range [0.0, 1.0].
      // RGB values may first need conversion (for example, dividing by 255 and then raising them to a power)
      // Where M for D65:	 0.4124564  0.3575761  0.1804375
      //0.2126729  0.7151522  0.0721750
      //0.0193339  0.1191920  0.9503041

      //// to make rgb values linear red, green, blue values
	  struct xyz XYZ;
      double rLinear = (double)R / 255.0;
      double gLinear = (double)G / 255.0;
      double bLinear = (double)B / 255.0;

      // convert to a sRGB form

      //double r =  Math.pow((rLinear ), 2.2) ;
      //double g =  Math.pow((gLinear ), 2.2) ;
      //double b = Math.pow((bLinear ), 2.2) ;
      double r, g, b;

      if (rLinear > 0.04045)
        r = pow(((rLinear + 0.055) / 1.055), 2.2);
      else
        r = rLinear / 12.92;

      if (gLinear > 0.04045)
        g = pow(((gLinear + 0.055) / 1.055), 2.2);
      else
        g = gLinear / 12.92;

      if (bLinear > 0.04045)
        b = pow(((bLinear + 0.055) / 1.055), 2.2);
      else
        b = bLinear / 12.92;


		XYZ.X=(r * 0.4124564 + g * 0.3575761 + b * 0.1804375);
        XYZ.Y=(r * 0.2126729 + g * 0.7151522 + b * 0.0721750);
        XYZ.Z=(r * 0.0193339 + g * 0.1191920 + b * 0.9503041);

		return XYZ;

		/*
	if(X>0.9504)
		X=0.9504;
	else if (X<0)
		X=0;
	else
		X=X;
	
	if(X>1)
		Y=1
	else if (Y<0)
		Y=0;
	else
		Y=Y;

	if(Z>1.0888)
		Z=1.0888;
	else if (Z<0)
		Z=0;
	else
		Z=Z;
*/
    }

__device__  double FX(double e)
    {
	
		if(e > 0.008856) 
			e=pow(e, (1/3));
		else
			e=((903.3 * e) + 16) / 116;
			return e;

		
    }

__device__ double Lxyz(double e)
    {
      if (e > 0.008856) 
	  e= (116 * pow(e, (1 / 3))) - 16 ;
	  else
	  e= (903.3 * e);
	  return e;
    }

//rgb to lab
__device__ void ToLAB(int R,int G,int B )
    {
      double Fx, Fy, Fz;
     
      RGBToXYZ(R,G,B);

      double yr = Y / 1.0000;
      double xr = X /0.9504;
      double zr = Z / 1.0888;

      Fx = FX(xr);
      Fy = FX(yr);
      Fz = FX(zr);

      L = Lxyz(yr);
      A = 500 * (Fx - Fy);
      B = 200 * (Fy - Fz);

     
    }
__device__ struct LAB ToLAB_st(int R,int G,int B )
    {
	  struct LAB lab;
	  struct xyz XYZ;
      double Fx, Fy, Fz;
     
      XYZ=RGBToXYZ_St(R,G,B);

      double yr = XYZ.Y / 1.0000;
      double xr = XYZ.X /0.9504;
      double zr = XYZ.Z / 1.0888;

      Fx = FX(xr);
      Fy = FX(yr);
      Fz = FX(zr);

      lab.L = Lxyz(yr);
      lab.A = 500 * (Fx - Fy);
      lab.B = 200 * (Fy - Fz);
	  return lab;
	       
    }

__device__ void XYZtoLAB(double X1, double Y1,double Z1)
{
	  double Fx, Fy, Fz;
           
      double yr = Y1 / 1.0000;
      double xr = X1 /0.9504;
      double zr = Z1 / 1.0888;

      Fx = FX(xr);
      Fy = FX(yr);
      Fz = FX(zr);

      L = Lxyz(yr);
      A = 500 * (Fx - Fy);
      B = 200 * (Fy - Fz);
}
__device__ struct LAB XYZtoLAB_st(double X1, double Y1,double Z1)
{
	  struct LAB lab;
	  double Fx, Fy, Fz;
           
      double yr =(double) Y1; /// 1.0000;
      double xr = (double)X1 /0.9504;
      double zr = (double)Z1 / 1.0888;

      Fx = FX(xr);
      Fy = FX(yr);
      Fz = FX(zr);

      lab.L = Lxyz(yr);
      lab.A = 500 * (Fx - Fy);
      lab.B = 200 * (Fy - Fz);

	  return lab;
}

__device__ void LABToXYZ(double L1,double A1,double B1)
    {
      double  xr, yr, zr, Xr, Yr, Zr, Fx, Fy, Fz;

      Fy = (L1 + 16) / 116;

      Fx = (A1 / 500) + Fy;

      Fz = Fy - (B1 / 200);

      if (pow(Fx, 3) > (0.008856)) 
		xr=pow(Fx, 3);
	  else
	    xr=((116 * Fx - 16) / 903.3);

      if(L1 > (0.008856 * 903.3))
		yr= pow(Fy, 3) ; 
	  else
		yr=L1 / 903.3;

      if (pow(Fz, 3) > (0.008856)) 
		zr= pow(Fz, 3);
	  else
		zr= ((116 * Fz - 16) / 903.3);

      //for D65
      Xr = 0.9504;

      Yr = 1;

      Zr = 1.0888;

      X = xr * Xr;
      Y = yr * Yr;
      Z = zr * Zr;

	/*
	if(X>0.9504)
		X=0.9504;
	else if (X<0)
		X=0;
	else
		X=X;
	
	if(X>1)
		Y=1
	else if (Y<0)
		Y=0;
	else
		Y=Y;

	if(Z>1.0888)
		Z=1.0888;
	else if (Z<0)
		Z=0;
	else
		Z=Z;
*/
	  }
__device__  struct xyz  LABToXYZ_st(double L1,double A1,double B1)
    {
	  struct xyz XYZ;
      double  xr, yr, zr, Xr, Yr, Zr, Fx, Fy, Fz;

      Fy = (L1 + 16) / 116;

      Fx = (A1 / 500) + Fy;

      Fz = Fy - (B1 / 200);

      if (pow(Fx, 3) > (0.008856)) 
		xr=pow(Fx, 3);
	  else
	    xr=((116 * Fx - 16) / 903.3);

      if(L1 > (0.008856 * 903.3))
		yr= pow(Fy, 3) ; 
	  else
		yr=L1 / 903.3;

      if (pow(Fz, 3) > (0.008856)) 
		zr= pow(Fz, 3);
	  else
		zr= ((116 * Fz - 16) / 903.3);

      //for D65
      Xr = 0.9504;

      Yr = 1;

      Zr = 1.0888;

      XYZ.X = xr * Xr;
      XYZ.Y = yr * Yr;
      XYZ.Z = zr * Zr;
	  return XYZ;

	/*
	if(X>0.9504)
		X=0.9504;
	else if (X<0)
		X=0;
	else
		X=X;
	
	if(X>1)
		Y=1
	else if (Y<0)
		Y=0;
	else
		Y=Y;

	if(Z>1.0888)
		Z=1.0888;
	else if (Z<0)
		Z=0;
	else
		Z=Z;
*/
	  }
	
// __global__ void correct(int *block_frame, double *block_background, double *block_profile,char *a, int *b) 
//{
//	//a[threadIdx.x] += b[threadIdx.x];
//	//GPU_FRAME_DIMENSIONS
//	for(int pixel = 0 ; pixel < 1 ; pixel++)
//	{
//		int R = block_frame[3*pixel + 0];
//		int G = block_frame[3*pixel + 1];
//		int B = block_frame[3*pixel + 2];
//
//		double bgX = block_background[3*pixel + 0];
//		double bgY = block_background[3*pixel + 1];
//		double bgZ = block_background[3*pixel + 2];
//
//		/*double bgX1 = block_background[3*pixel + 0];
//		double bgY1 = block_background[3*pixel + 1];
//		double bgZ1 = block_background[3*pixel + 2];
//
//		block_frame[3*pixel + 0]=(int)(block_background[3*pixel + 0]*block_background[3*pixel + 0]);
//		block_frame[3*pixel + 1]=(int)(block_background[3*pixel + 1]*block_background[3*pixel + 1]);
//		block_frame[3*pixel + 2]=(int)(block_background[3*pixel + 2]*block_background[3*pixel + 2]);
//
//	}*/
//
//		
//		double FrameX,FrameY,FrameZ,FrameL,FrameA,FrameB,BlendL,BlendA,BlendB;
//		
//		
//		// to find the best fit bin color
//		// SAVES LAB in device global L,A,B variables
//		ToLAB(R,G,B);
//		FrameL=L;
//		FrameA=A;
//		FrameB=B;
//
//		double DistanceInBin =  DBL_MIN;
//		double ClosestBinDistance;
//		int BinIndex=0;
//		
//		double keyL;
//		double keyA;
//		double keyB;
//
//		for(int bin = 0 ; bin < PROFILE_SIZE ; bin++)
//		{			
//			 keyL = block_profile[6*bin + 0];
//			 keyA = block_profile[6*bin + 1];
//			 keyB = block_profile[6*bin + 2];
//
//			 ClosestBinDistance=distance(FrameL,FrameA,FrameB,keyL,keyA,keyB);
//
//			 if (DistanceInBin >= ClosestBinDistance)
//	               continue;
//				
//	          DistanceInBin = ClosestBinDistance;
//			  BinIndex=bin;
//		}
//
//		FrameL=block_profile[6*BinIndex + 3];
//		FrameA=block_profile[6*BinIndex + 4];
//		FrameB=block_profile[6*BinIndex + 5];
//
//		//DO YOUR MAGIC
//		DistanceInBin =  DBL_MIN;
//		ClosestBinDistance=0;
//		BinIndex=0;
//		for(int bin = 0 ; bin < PROFILE_SIZE ; bin++)
//		{			
//			
//			double valueL = block_profile[6*bin + 3];
//			double valueA = block_profile[6*bin + 4];
//			double valueB = block_profile[6*bin + 5];
//
//			//getting the xyz values of the chocen bin
//			LABToXYZ(valueL,valueA,valueB);
//
//			FrameX=X;
//			FrameY=Y;
//			FrameZ=Z;
//
//			addXYZ(FrameX,FrameY,FrameZ,bgX,bgY,bgZ);
//			
//			XYZtoLAB(X,Y,Z);
//			BlendL=L;
//			BlendA=A;
//			BlendB=B;
//
//
//			ClosestBinDistance=distance(FrameL,FrameA,FrameB,BlendL,BlendA,BlendB);
//
//			 if (DistanceInBin >= ClosestBinDistance)
//	               continue;
//				
//	             DistanceInBin = ClosestBinDistance;
//			  BinIndex=bin;
//
//		}
//
//		block_frame[3*pixel + 0]=(int)block_profile[6*BinIndex + 3];
//		block_frame[3*pixel + 1]=(int)block_profile[6*BinIndex + 4];
//		block_frame[3*pixel + 2]=(int)block_profile[6*BinIndex + 5];
//	
//	}
//}
// 
__global__ void correct2(int *block_frame, double *block_background, double *block_profile,char *a, int *b) 
{
	//a[threadIdx.x] += b[threadIdx.x];
	//GPU_FRAME_DIMENSIONS
	for(int pixel = 0 ; pixel <1 ; pixel++)
	{
		 struct xyz XYZ;
		 struct xyz XYZ_blend;
		 struct LAB lab;
		 struct LAB Keylab;
		 struct LAB lab_blend;

		int R = block_frame[3*pixel + 0];
		int G = block_frame[3*pixel + 1];
		int B = block_frame[3*pixel + 2];

		double bgX = block_background[3*pixel + 0];
		double bgY = block_background[3*pixel + 1];
		double bgZ = block_background[3*pixel + 2];

		/*double bgX1 = block_background[3*pixel + 0];
		double bgY1 = block_background[3*pixel + 1];
		double bgZ1 = block_background[3*pixel + 2];

		block_frame[3*pixel + 0]=(int)(block_background[3*pixel + 0]*block_background[3*pixel + 0]);
		block_frame[3*pixel + 1]=(int)(block_background[3*pixel + 1]*block_background[3*pixel + 1]);
		block_frame[3*pixel + 2]=(int)(block_background[3*pixel + 2]*block_background[3*pixel + 2]);

	}*/

		
		//double FrameX,FrameY,FrameZ,FrameL,FrameA,FrameB,BlendL,BlendA,BlendB;
		
		
		// to find the best fit bin color
		// SAVES LAB in device global L,A,B variables
		lab=ToLAB_st(R,G,B);
		

		double DistanceInBin =  DBL_MAX;
		double ClosestBinDistance;
		int BinIndex=0;
			

		for(int bin = 0 ; bin < 8376 ; bin++)
		{			
			 Keylab.L = block_profile[6*bin + 0];
			 Keylab.A = block_profile[6*bin + 1];
			 Keylab.B = block_profile[6*bin + 2];

			 ClosestBinDistance=distance(lab.L,lab.A,lab.B, Keylab.L, Keylab.A, Keylab.B);

			 if (DistanceInBin >= ClosestBinDistance)
	               continue;
				
	          DistanceInBin = ClosestBinDistance;
			  BinIndex=bin;
		}

		lab.L=block_profile[6*BinIndex + 3];
		lab.A=block_profile[6*BinIndex + 4];
		lab.B=block_profile[6*BinIndex + 5];

		//DO YOUR MAGIC
		DistanceInBin =  DBL_MAX;
		ClosestBinDistance=0;
		BinIndex=0;
		for(int bin = 0 ; bin < 8376 ; bin++)
		{			
			
			double valueL = block_profile[6*bin + 3];
			double valueA = block_profile[6*bin + 4];
			double valueB = block_profile[6*bin + 5];

			//getting the xyz values of the chocen bin
			XYZ=LABToXYZ_st(valueL,valueA,valueB);


			XYZ_blend=addXYZ_st(XYZ.X,XYZ.Y,XYZ.Z,bgX,bgY,bgZ);
			
			lab_blend=XYZtoLAB_st(XYZ_blend.X,XYZ_blend.Y,XYZ_blend.Z);
			
			ClosestBinDistance=distance(lab.L,lab.A,lab.B,lab_blend.L,lab_blend.A,lab_blend.B);

			 if (DistanceInBin >= ClosestBinDistance)
	               continue;
				
	             DistanceInBin = ClosestBinDistance;
			  BinIndex=bin;

		}

		block_frame[3*pixel + 0]=(int)block_profile[6*BinIndex + 3];
		block_frame[3*pixel + 1]=(int)block_profile[6*BinIndex + 4];
		block_frame[3*pixel + 2]=(int)block_profile[6*BinIndex + 5];
	
	}
}

__global__ void correct3(int *block_frame, double *block_background, double *block_profile) 
{
	int pixel = 0;
	int block_background_index = 3 * pixel;

	double blendX;
	double blendY;
	double blendZ;

	double tempX;
	double tempY;
	double tempZ;

	double bgX = block_background[block_background_index + 0];
	double bgY = block_background[block_background_index + 1];
	double bgZ = block_background[block_background_index + 2];

	double closestColor =  DBL_MAX;
	double closestBinDistance;
	int binIndex=0;	
	int block_profile_index = 0;
	double diffX, diffY, diffZ, result;

	for(int bin = 0 ; bin < 8000; bin++)
	{			
		block_profile_index = 6*bin;
		tempX = block_profile[block_profile_index + 3];
		tempY = block_profile[block_profile_index + 4];
		tempZ = block_profile[block_profile_index + 5];

		//getting the xyz values of the chocen bin

		/*blendX = tempX + bgX;
		blendY = tempY + bgY;
		blendZ = tempZ + bgZ;*/
		{
			diffX = blendX - 0;
			diffY = blendY - 0;
			diffZ = blendZ - 0;
			diffX = diffX * diffX;
			diffY = diffY * diffY;
			diffZ = diffZ * diffZ;
			closestBinDistance = sqrt(diffX + diffY + diffZ);
		}

		if (closestBinDistance >= closestColor)
			continue;

		closestColor = closestBinDistance;
		binIndex = bin;
	}

	block_profile_index = 6*binIndex;
	block_frame[block_background_index + 0]= (int)block_profile[block_profile_index + 3];
	block_frame[block_background_index + 1]= (int)block_profile[block_profile_index + 4];
	block_frame[block_background_index + 2]= (int)block_profile[block_profile_index + 5];
}

//basic cuda whihc runs on a single thread 
int main(int argc, char** argv)
{
	clock_t tstart;
	clock_t end;
	double runTime;

	double *background,*profile,*partitioned_background;
	int *frame,*partitioned_frame;

	//var display-profile -- lookup table in LAB
	profile = (double*) malloc(PROFILE_ARRAY_SIZE * sizeof(double));

	for(int index = 0 ; index < PROFILE_ARRAY_SIZE ; index++)
		profile[index] = 30;

	//var frame-image -- XYZ
	frame =	(int*)malloc(FRAME_ARRAY_SIZE * sizeof(int));
	for(int index = 0 ; index < FRAME_ARRAY_SIZE ; index++)
		frame[index] = 1;
	
	//var background-image -- background image in XYZ
	background= (double*)malloc(FRAME_ARRAY_SIZE * sizeof(double));
	for(int index = 0 ; index < FRAME_ARRAY_SIZE ; index++)
		background[index] = 0;

	//broken frame 
	partitioned_frame =	(int*)malloc( PARTITIONED_FRAME_DIMENSIONS* sizeof(int));
	
	
	///broken background-image 
	partitioned_background= (double*)malloc(PARTITIONED_FRAME_DIMENSIONS * sizeof(double));
	

	//pointers on the device
	double *gpu_profile;
	int *gpu_frame;
	double *gpu_background;

	printf("prg starting\n");

	const int psize = PROFILE_ARRAY_SIZE * sizeof(double);
	const int fsize = FRAME_ARRAY_SIZE * sizeof(int);
	const int bgsize = FRAME_ARRAY_SIZE * sizeof(double);
	
	//memory allocation on the GPU
	hipMalloc(&gpu_profile, PROFILE_ARRAY_SIZE * sizeof(double)); 
	hipMalloc(&gpu_frame, FRAME_ARRAY_SIZE * sizeof(int)); 
	hipMalloc(&gpu_background, FRAME_ARRAY_SIZE * sizeof(double)); 

	//1- copy the profile
	hipMemcpy(gpu_profile, profile, psize, hipMemcpyHostToDevice); 

	// start outer timer
	tstart = clock();
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	int partition_index=0;
	//image partitioned into 10 parts
	for(int f = 0 ; f < FRAME_DIMENSIONS ; f=f+PARTITIONED_FRAME_DIMENSIONS)
	{
		// assigning partitioed values of the frame and background
		for(int index = f ; index <  f+PARTITIONED_FRAME_DIMENSIONS ; index++)
		{

			partitioned_frame[partition_index] = frame[index];
			partitioned_background[partition_index] = background[index];
			partition_index++;
		}
		partition_index=0;
				

		//2- pass the image to correct to the GPU
		//cudaMemcpy(gpu_frame, frame, FRAME_ARRAY_SIZE * sizeof(int), cudaMemcpyHostToDevice); 
		hipMemcpy(gpu_frame, partitioned_frame, PARTITIONED_FRAME_DIMENSIONS * sizeof(int), hipMemcpyHostToDevice); 
		//3- pass the background image to the GPU
		//cudaMemcpy(gpu_background, background, FRAME_ARRAY_SIZE * sizeof(double), cudaMemcpyHostToDevice); 
		hipMemcpy(gpu_background, partitioned_background, PARTITIONED_FRAME_DIMENSIONS * sizeof(double), hipMemcpyHostToDevice); 
		
		dim3 threadsPerBlock(32, 32);
		dim3 numBlocks(PART_FRAME_WIDTH/threadsPerBlock.x, PART_FRAME_HEIGHT/threadsPerBlock.y); 
		
		// Start record
		hipEventRecord(start, NULL);
		correct3<<<numBlocks, threadsPerBlock>>>(gpu_frame, gpu_background, gpu_profile);
		hipMemcpy(frame, gpu_frame, FRAME_ARRAY_SIZE * sizeof(int), hipMemcpyDeviceToHost);
		hipEventRecord(stop, NULL);
		hipEventSynchronize(stop);
		float elapsedTime=-1;
		hipEventElapsedTime(&elapsedTime, start, stop);
		printf("Run time is: %f \n",elapsedTime);

	}


	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	for(int index = 0 ; index < 3 ; index++)
		printf("%d\n", frame[index]);
	
	end = clock();
	runTime = (end-tstart);
	printf("total Run time is %g mil;liseconds \n",runTime);
	hipFree( gpu_profile );
	hipFree( gpu_frame );
	hipFree( gpu_background );

	free(profile);
	free(frame );
	free(background );

	return EXIT_SUCCESS;
}

