#include <stdio.h>
#include <time.h>
#include <math.h>
#include <float.h>

#include "hip/hip_runtime.h"


 
const int blocksize = 800; 
const int N = 16; 


const int PROFILE_SIZE = 8376;
const int PROFILE_ARRAY_SIZE = PROFILE_SIZE * 6;
__device__ const int GPU_PROFILE_SIZE = 8376;

const int FRAME_DIMENSIONS = 800 * 600;
const int FRAME_ARRAY_SIZE = FRAME_DIMENSIONS * 3;
__device__ const int GPU_FRAME_DIMENSIONS = 800 * 600;

__device__ double X,Y,Z,L,A,B;
__device__ struct xyz
{
	 double X;
	 double Y;
	 double Z;
};

__device__ struct LAB
{
	 double L;
	 double A;
	 double B;
};
__device__ double distance(double L1,double A1,double B1,double L2,double A2,double B2)
{
	 double l, a, b, result,sqresult;
      l = L1 - L2;
      a = A1 - A2;
      b = B1 - B2;
      l = l * l;
      a = a * a;
      b = b * b;
	  result = l + a + b;

	  sqresult = sqrt(result);
	  return sqresult;
}

__device__ void addXYZ(double X1,double Y1,double Z1,double X2,double Y2,double Z2)
{
	X=X1+X2;
	Y=Y1+Y2;
	Z=Z1+Z2;
/*
	if(X>0.9504)
		X=0.9504;
	else if (X<0)
		X=0;
	else
		X=X;
	
	if(X>1)
		Y=1
	else if (Y<0)
		Y=0;
	else
		Y=Y;

	if(Z>1.0888)
		Z=1.0888;
	else if (Z<0)
		Z=0;
	else
		Z=Z;
*/
}
__device__  struct xyz addXYZ_st(double X1,double Y1,double Z1,double X2,double Y2,double Z2)
{
	 struct xyz XYZ;
	XYZ.X=X1+X2;
	XYZ.Y=Y1+Y2;
	XYZ.Z=Z1+Z2;
	return XYZ;
/*
	if(X>0.9504)
		X=0.9504;
	else if (X<0)
		X=0;
	else
		X=X;
	
	if(X>1)
		Y=1
	else if (Y<0)
		Y=0;
	else
		Y=Y;

	if(Z>1.0888)
		Z=1.0888;
	else if (Z<0)
		Z=0;
	else
		Z=Z;
*/
}

// rgb to xyz
//__device__ void RGBToXYZ(unsigned char R,unsigned char G,unsigned char B )
__device__ void RGBToXYZ(int R,int G,int B )
    {
      // by the formula given the the web page http://www.brucelindbloom.com/index.html [XYZ]=[M][RGB]
      //In order to properly use this matrix, the RGB values must be linear and in the nominal range [0.0, 1.0].
      // RGB values may first need conversion (for example, dividing by 255 and then raising them to a power)
      // Where M for D65:	 0.4124564  0.3575761  0.1804375
      //0.2126729  0.7151522  0.0721750
      //0.0193339  0.1191920  0.9503041

      //// to make rgb values linear red, green, blue values
      double rLinear = (double)R / 255.0;
      double gLinear = (double)G / 255.0;
      double bLinear = (double)B / 255.0;

      // convert to a sRGB form

      //double r =  Math.pow((rLinear ), 2.2) ;
      //double g =  Math.pow((gLinear ), 2.2) ;
      //double b = Math.pow((bLinear ), 2.2) ;
      double r, g, b;

      if (rLinear > 0.04045)
        r = pow(((rLinear + 0.055) / 1.055), 2.2);
      else
        r = rLinear / 12.92;

      if (gLinear > 0.04045)
        g = pow(((gLinear + 0.055) / 1.055), 2.2);
      else
        g = gLinear / 12.92;

      if (bLinear > 0.04045)
        b = pow(((bLinear + 0.055) / 1.055), 2.2);
      else
        b = bLinear / 12.92;


		X=(r * 0.4124564 + g * 0.3575761 + b * 0.1804375);
        Y=(r * 0.2126729 + g * 0.7151522 + b * 0.0721750);
        Z=(r * 0.0193339 + g * 0.1191920 + b * 0.9503041);

		/*
	if(X>0.9504)
		X=0.9504;
	else if (X<0)
		X=0;
	else
		X=X;
	
	if(X>1)
		Y=1
	else if (Y<0)
		Y=0;
	else
		Y=Y;

	if(Z>1.0888)
		Z=1.0888;
	else if (Z<0)
		Z=0;
	else
		Z=Z;
*/
    }
__device__ struct xyz RGBToXYZ_St(int R,int G,int B )
    {
      // by the formula given the the web page http://www.brucelindbloom.com/index.html [XYZ]=[M][RGB]
      //In order to properly use this matrix, the RGB values must be linear and in the nominal range [0.0, 1.0].
      // RGB values may first need conversion (for example, dividing by 255 and then raising them to a power)
      // Where M for D65:	 0.4124564  0.3575761  0.1804375
      //0.2126729  0.7151522  0.0721750
      //0.0193339  0.1191920  0.9503041

      //// to make rgb values linear red, green, blue values
	  struct xyz XYZ;
      double rLinear = (double)R / 255.0;
      double gLinear = (double)G / 255.0;
      double bLinear = (double)B / 255.0;

      // convert to a sRGB form

      //double r =  Math.pow((rLinear ), 2.2) ;
      //double g =  Math.pow((gLinear ), 2.2) ;
      //double b = Math.pow((bLinear ), 2.2) ;
      double r, g, b;

      if (rLinear > 0.04045)
        r = pow(((rLinear + 0.055) / 1.055), 2.2);
      else
        r = rLinear / 12.92;

      if (gLinear > 0.04045)
        g = pow(((gLinear + 0.055) / 1.055), 2.2);
      else
        g = gLinear / 12.92;

      if (bLinear > 0.04045)
        b = pow(((bLinear + 0.055) / 1.055), 2.2);
      else
        b = bLinear / 12.92;


		XYZ.X=(r * 0.4124564 + g * 0.3575761 + b * 0.1804375);
        XYZ.Y=(r * 0.2126729 + g * 0.7151522 + b * 0.0721750);
        XYZ.Z=(r * 0.0193339 + g * 0.1191920 + b * 0.9503041);

		return XYZ;

		/*
	if(X>0.9504)
		X=0.9504;
	else if (X<0)
		X=0;
	else
		X=X;
	
	if(X>1)
		Y=1
	else if (Y<0)
		Y=0;
	else
		Y=Y;

	if(Z>1.0888)
		Z=1.0888;
	else if (Z<0)
		Z=0;
	else
		Z=Z;
*/
    }

__device__  double FX(double e)
    {
	
		if(e > 0.008856) 
			e=pow(e, (1/3));
		else
			e=((903.3 * e) + 16) / 116;
			return e;

		
    }

__device__ double Lxyz(double e)
    {
      if (e > 0.008856) 
	  e= (116 * pow(e, (1 / 3))) - 16 ;
	  else
	  e= (903.3 * e);
	  return e;
    }

//rgb to lab
__device__ void ToLAB(int R,int G,int B )
    {
      double Fx, Fy, Fz;
     
      RGBToXYZ(R,G,B);

      double yr = Y / 1.0000;
      double xr = X /0.9504;
      double zr = Z / 1.0888;

      Fx = FX(xr);
      Fy = FX(yr);
      Fz = FX(zr);

      L = Lxyz(yr);
      A = 500 * (Fx - Fy);
      B = 200 * (Fy - Fz);

     
    }
__device__ struct LAB ToLAB_st(int R,int G,int B )
    {
	  struct LAB lab;
	  struct xyz XYZ;
      double Fx, Fy, Fz;
     
      XYZ=RGBToXYZ_St(R,G,B);

      double yr = XYZ.Y / 1.0000;
      double xr = XYZ.X /0.9504;
      double zr = XYZ.Z / 1.0888;

      Fx = FX(xr);
      Fy = FX(yr);
      Fz = FX(zr);

      lab.L = Lxyz(yr);
      lab.A = 500 * (Fx - Fy);
      lab.B = 200 * (Fy - Fz);
	  return lab;
	       
    }

__device__ void XYZtoLAB(double X1, double Y1,double Z1)
{
	  double Fx, Fy, Fz;
           
      double yr = Y1 / 1.0000;
      double xr = X1 /0.9504;
      double zr = Z1 / 1.0888;

      Fx = FX(xr);
      Fy = FX(yr);
      Fz = FX(zr);

      L = Lxyz(yr);
      A = 500 * (Fx - Fy);
      B = 200 * (Fy - Fz);
}
__device__ struct LAB XYZtoLAB_st(double X1, double Y1,double Z1)
{
	  struct LAB lab;
	  double Fx, Fy, Fz;
           
      double yr =(double) Y1; /// 1.0000;
      double xr = (double)X1 /0.9504;
      double zr = (double)Z1 / 1.0888;

      Fx = FX(xr);
      Fy = FX(yr);
      Fz = FX(zr);

      lab.L = Lxyz(yr);
      lab.A = 500 * (Fx - Fy);
      lab.B = 200 * (Fy - Fz);

	  return lab;
}

__device__ void LABToXYZ(double L1,double A1,double B1)
    {
      double  xr, yr, zr, Xr, Yr, Zr, Fx, Fy, Fz;

      Fy = (L1 + 16) / 116;

      Fx = (A1 / 500) + Fy;

      Fz = Fy - (B1 / 200);

      if (pow(Fx, 3) > (0.008856)) 
		xr=pow(Fx, 3);
	  else
	    xr=((116 * Fx - 16) / 903.3);

      if(L1 > (0.008856 * 903.3))
		yr= pow(Fy, 3) ; 
	  else
		yr=L1 / 903.3;

      if (pow(Fz, 3) > (0.008856)) 
		zr= pow(Fz, 3);
	  else
		zr= ((116 * Fz - 16) / 903.3);

      //for D65
      Xr = 0.9504;

      Yr = 1;

      Zr = 1.0888;

      X = xr * Xr;
      Y = yr * Yr;
      Z = zr * Zr;

	/*
	if(X>0.9504)
		X=0.9504;
	else if (X<0)
		X=0;
	else
		X=X;
	
	if(X>1)
		Y=1
	else if (Y<0)
		Y=0;
	else
		Y=Y;

	if(Z>1.0888)
		Z=1.0888;
	else if (Z<0)
		Z=0;
	else
		Z=Z;
*/
	  }
__device__  struct xyz  LABToXYZ_st(double L1,double A1,double B1)
    {
	  struct xyz XYZ;
      double  xr, yr, zr, Xr, Yr, Zr, Fx, Fy, Fz;

      Fy = (L1 + 16) / 116;

      Fx = (A1 / 500) + Fy;

      Fz = Fy - (B1 / 200);

      if (pow(Fx, 3) > (0.008856)) 
		xr=pow(Fx, 3);
	  else
	    xr=((116 * Fx - 16) / 903.3);

      if(L1 > (0.008856 * 903.3))
		yr= pow(Fy, 3) ; 
	  else
		yr=L1 / 903.3;

      if (pow(Fz, 3) > (0.008856)) 
		zr= pow(Fz, 3);
	  else
		zr= ((116 * Fz - 16) / 903.3);

      //for D65
      Xr = 0.9504;

      Yr = 1;

      Zr = 1.0888;

      XYZ.X = xr * Xr;
      XYZ.Y = yr * Yr;
      XYZ.Z = zr * Zr;
	  return XYZ;

	/*
	if(X>0.9504)
		X=0.9504;
	else if (X<0)
		X=0;
	else
		X=X;
	
	if(X>1)
		Y=1
	else if (Y<0)
		Y=0;
	else
		Y=Y;

	if(Z>1.0888)
		Z=1.0888;
	else if (Z<0)
		Z=0;
	else
		Z=Z;
*/
	  }
	
// __global__ void correct(int *block_frame, double *block_background, double *block_profile,char *a, int *b) 
//{
//	//a[threadIdx.x] += b[threadIdx.x];
//	//GPU_FRAME_DIMENSIONS
//	for(int pixel = 0 ; pixel < 1 ; pixel++)
//	{
//		int R = block_frame[3*pixel + 0];
//		int G = block_frame[3*pixel + 1];
//		int B = block_frame[3*pixel + 2];
//
//		double bgX = block_background[3*pixel + 0];
//		double bgY = block_background[3*pixel + 1];
//		double bgZ = block_background[3*pixel + 2];
//
//		/*double bgX1 = block_background[3*pixel + 0];
//		double bgY1 = block_background[3*pixel + 1];
//		double bgZ1 = block_background[3*pixel + 2];
//
//		block_frame[3*pixel + 0]=(int)(block_background[3*pixel + 0]*block_background[3*pixel + 0]);
//		block_frame[3*pixel + 1]=(int)(block_background[3*pixel + 1]*block_background[3*pixel + 1]);
//		block_frame[3*pixel + 2]=(int)(block_background[3*pixel + 2]*block_background[3*pixel + 2]);
//
//	}*/
//
//		
//		double FrameX,FrameY,FrameZ,FrameL,FrameA,FrameB,BlendL,BlendA,BlendB;
//		
//		
//		// to find the best fit bin color
//		// SAVES LAB in device global L,A,B variables
//		ToLAB(R,G,B);
//		FrameL=L;
//		FrameA=A;
//		FrameB=B;
//
//		double DistanceInBin =  DBL_MIN;
//		double ClosestBinDistance;
//		int BinIndex=0;
//		
//		double keyL;
//		double keyA;
//		double keyB;
//
//		for(int bin = 0 ; bin < PROFILE_SIZE ; bin++)
//		{			
//			 keyL = block_profile[6*bin + 0];
//			 keyA = block_profile[6*bin + 1];
//			 keyB = block_profile[6*bin + 2];
//
//			 ClosestBinDistance=distance(FrameL,FrameA,FrameB,keyL,keyA,keyB);
//
//			 if (DistanceInBin >= ClosestBinDistance)
//	               continue;
//				
//	          DistanceInBin = ClosestBinDistance;
//			  BinIndex=bin;
//		}
//
//		FrameL=block_profile[6*BinIndex + 3];
//		FrameA=block_profile[6*BinIndex + 4];
//		FrameB=block_profile[6*BinIndex + 5];
//
//		//DO YOUR MAGIC
//		DistanceInBin =  DBL_MIN;
//		ClosestBinDistance=0;
//		BinIndex=0;
//		for(int bin = 0 ; bin < PROFILE_SIZE ; bin++)
//		{			
//			
//			double valueL = block_profile[6*bin + 3];
//			double valueA = block_profile[6*bin + 4];
//			double valueB = block_profile[6*bin + 5];
//
//			//getting the xyz values of the chocen bin
//			LABToXYZ(valueL,valueA,valueB);
//
//			FrameX=X;
//			FrameY=Y;
//			FrameZ=Z;
//
//			addXYZ(FrameX,FrameY,FrameZ,bgX,bgY,bgZ);
//			
//			XYZtoLAB(X,Y,Z);
//			BlendL=L;
//			BlendA=A;
//			BlendB=B;
//
//
//			ClosestBinDistance=distance(FrameL,FrameA,FrameB,BlendL,BlendA,BlendB);
//
//			 if (DistanceInBin >= ClosestBinDistance)
//	               continue;
//				
//	             DistanceInBin = ClosestBinDistance;
//			  BinIndex=bin;
//
//		}
//
//		block_frame[3*pixel + 0]=(int)block_profile[6*BinIndex + 3];
//		block_frame[3*pixel + 1]=(int)block_profile[6*BinIndex + 4];
//		block_frame[3*pixel + 2]=(int)block_profile[6*BinIndex + 5];
//	
//	}
//}
// 
__global__ void correct2(int *block_frame, double *block_background, double *block_profile,char *a, int *b) 
{
	//a[threadIdx.x] += b[threadIdx.x];
	//GPU_FRAME_DIMENSIONS
	for(int pixel = 0 ; pixel <1 ; pixel++)
	{
		 struct xyz XYZ;
		 struct xyz XYZ_blend;
		 struct LAB lab;
		 struct LAB Keylab;
		 struct LAB lab_blend;

		int R = block_frame[3*pixel + 0];
		int G = block_frame[3*pixel + 1];
		int B = block_frame[3*pixel + 2];

		double bgX = block_background[3*pixel + 0];
		double bgY = block_background[3*pixel + 1];
		double bgZ = block_background[3*pixel + 2];

		/*double bgX1 = block_background[3*pixel + 0];
		double bgY1 = block_background[3*pixel + 1];
		double bgZ1 = block_background[3*pixel + 2];

		block_frame[3*pixel + 0]=(int)(block_background[3*pixel + 0]*block_background[3*pixel + 0]);
		block_frame[3*pixel + 1]=(int)(block_background[3*pixel + 1]*block_background[3*pixel + 1]);
		block_frame[3*pixel + 2]=(int)(block_background[3*pixel + 2]*block_background[3*pixel + 2]);

	}*/

		
		//double FrameX,FrameY,FrameZ,FrameL,FrameA,FrameB,BlendL,BlendA,BlendB;
		
		
		// to find the best fit bin color
		// SAVES LAB in device global L,A,B variables
		lab=ToLAB_st(R,G,B);
		

		double DistanceInBin =  DBL_MAX;
		double ClosestBinDistance;
		int BinIndex=0;
			

		for(int bin = 0 ; bin < 8376 ; bin++)
		{			
			 Keylab.L = block_profile[6*bin + 0];
			 Keylab.A = block_profile[6*bin + 1];
			 Keylab.B = block_profile[6*bin + 2];

			 ClosestBinDistance=distance(lab.L,lab.A,lab.B, Keylab.L, Keylab.A, Keylab.B);

			 if (DistanceInBin >= ClosestBinDistance)
	               continue;
				
	          DistanceInBin = ClosestBinDistance;
			  BinIndex=bin;
		}

		lab.L=block_profile[6*BinIndex + 3];
		lab.A=block_profile[6*BinIndex + 4];
		lab.B=block_profile[6*BinIndex + 5];

		//DO YOUR MAGIC
		DistanceInBin =  DBL_MAX;
		ClosestBinDistance=0;
		BinIndex=0;
		for(int bin = 0 ; bin < 8376 ; bin++)
		{			
			
			double valueL = block_profile[6*bin + 3];
			double valueA = block_profile[6*bin + 4];
			double valueB = block_profile[6*bin + 5];

			//getting the xyz values of the chocen bin
			XYZ=LABToXYZ_st(valueL,valueA,valueB);


			XYZ_blend=addXYZ_st(XYZ.X,XYZ.Y,XYZ.Z,bgX,bgY,bgZ);
			
			lab_blend=XYZtoLAB_st(XYZ_blend.X,XYZ_blend.Y,XYZ_blend.Z);
			
			ClosestBinDistance=distance(lab.L,lab.A,lab.B,lab_blend.L,lab_blend.A,lab_blend.B);

			 if (DistanceInBin >= ClosestBinDistance)
	               continue;
				
	             DistanceInBin = ClosestBinDistance;
			  BinIndex=bin;

		}

		block_frame[3*pixel + 0]=(int)block_profile[6*BinIndex + 3];
		block_frame[3*pixel + 1]=(int)block_profile[6*BinIndex + 4];
		block_frame[3*pixel + 2]=(int)block_profile[6*BinIndex + 5];
	
	}
}

__global__ void correct3(int *block_frame, double *block_background, double *block_profile) 
{
	//a[threadIdx.x] += b[threadIdx.x];
	//GPU_FRAME_DIMENSIONS
	for(int pixel = 0 ; pixel <1 ; pixel++)
	{
		 //struct xyz XYZ;
		 struct xyz XYZ_blend;
		 struct LAB lab;
		// struct LAB Keylab;
		 struct LAB lab_blend;

		
		double bgX = block_background[3*pixel + 0];
		double bgY = block_background[3*pixel + 1];
		double bgZ = block_background[3*pixel + 2];

		
		double closestColor =  DBL_MAX;
		double ClosestBinDistance;
		int BinIndex=0;

		lab.L=block_profile[6*BinIndex + 3];
		lab.A=block_profile[6*BinIndex + 4];
		lab.B=block_profile[6*BinIndex + 5];

		//DO YOUR MAGIC
		
		
			
		for(int bin = 0 ; bin < 8376 ; bin++)
		{			
			
			double valueL = block_profile[6*bin + 3];
			double valueA = block_profile[6*bin + 4];
			double valueB = block_profile[6*bin + 5];

			//getting the xyz values of the chocen bin
			
			XYZ_blend=addXYZ_st(0,0,0,bgX,bgY,bgZ);
			
			lab_blend=XYZtoLAB_st(XYZ_blend.X,XYZ_blend.Y,XYZ_blend.Z);
			
			ClosestBinDistance=distance(lab.L,lab.A,lab.B,lab_blend.L,lab_blend.A,lab_blend.B);

			 if (ClosestBinDistance >= closestColor)
	               continue;
				
	          closestColor = ClosestBinDistance;
			  BinIndex=bin;

		}

		block_frame[3*pixel + 0]=(int)block_profile[6*BinIndex + 3];
		block_frame[3*pixel + 1]=(int)block_profile[6*BinIndex + 4];
		block_frame[3*pixel + 2]=(int)block_profile[6*BinIndex + 5];
	
	}
}

//basic cuda whihc runs on a single thread 
int main(int argc, char** argv)
{
	//
	char a[N] = "Hello \0\0\0\0\0\0";
	int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

	char *ad;
	int *bd;
	const int csize = N*sizeof(char);
	const int isize = N*sizeof(int);
 
	printf("%s", a);

	///0
	clock_t tstart;
	clock_t end;
	double runTime;

	double *background,*profile;
	//unsigned char *frame;
	int *frame;

	//var display-profile -- lookup table in LAB
	profile = (double*) malloc(PROFILE_ARRAY_SIZE * sizeof(double));

	for(int index = 0 ; index < PROFILE_ARRAY_SIZE ; index++)
		profile[index] = 30;

	//var frame-image -- image to correct in RGB
	/*frame=	(unsigned char*)malloc(FRAME_ARRAY_SIZE * sizeof(unsigned char));
	for(int index = 0 ; index < FRAME_ARRAY_SIZE ; index++)
		frame[index] = '3';
	*/

	frame =	(int*)malloc(FRAME_ARRAY_SIZE * sizeof(int));
	for(int index = 0 ; index < FRAME_ARRAY_SIZE ; index++)
		frame[index] = 1;
	//var background-image -- background image in YXZ
	background= (double*)malloc(FRAME_ARRAY_SIZE * sizeof(double));
	for(int index = 0 ; index < FRAME_ARRAY_SIZE ; index++)
		background[index] = 0;

	//pointers on the device
	double *gpu_profile;
	//unsigned char *gpu_frame;
	int *gpu_frame;

	double *gpu_background;
	printf("prg starting\n");

	const int psize = PROFILE_ARRAY_SIZE * sizeof(double);
	//const int fsize = FRAME_ARRAY_SIZE * sizeof(unsigned char);
	const int fsize = FRAME_ARRAY_SIZE * sizeof(int);
	const int bgsize = FRAME_ARRAY_SIZE * sizeof(double);
	
	//memory allocation on the GPU
	hipMalloc(&gpu_profile, PROFILE_ARRAY_SIZE * sizeof(double)); 
	//cudaMalloc(&gpu_frame, FRAME_ARRAY_SIZE * sizeof(unsigned char)); 
	hipMalloc(&gpu_frame, FRAME_ARRAY_SIZE * sizeof(int)); 
	hipMalloc(&gpu_background, FRAME_ARRAY_SIZE * sizeof(double)); 

	//1- copy the profile
	hipMemcpy(gpu_profile, profile, psize, hipMemcpyHostToDevice); 

	// start outer timer
	tstart = clock();
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	//-- for 100 frames
	for(int f = 0 ; f < 1 ; f++)
	{
		//start inner timer
		
				
		//2- pass the image to correct to the GPU
		
		hipMemcpy(gpu_frame, frame, FRAME_ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice); 

		//3- pass the background image to the GPU
		hipMemcpy(gpu_background, background, FRAME_ARRAY_SIZE * sizeof(double), hipMemcpyHostToDevice); 
		
		
		//4- call the kernel
		dim3 block_size(20,20);
		dim3 dimGrid( 1, 1 );

		dim3 grid_size;
        grid_size.x = (100)/block_size.x;  /*< Greater than or equal to image width */
		grid_size.y = (100)/block_size.y;
	
		
				// Start record
		hipEventRecord(start, NULL);
		correct3<<<grid_size, block_size>>>(gpu_frame, gpu_background,gpu_profile);
		hipMemcpy(frame, gpu_frame, FRAME_ARRAY_SIZE * sizeof(int), hipMemcpyDeviceToHost); 
			hipEventRecord(stop, NULL);
			hipEventSynchronize(stop);
			float elapsedTime=-1;
			hipEventElapsedTime(&elapsedTime, start, stop);
			printf("Run time is: %f \n",elapsedTime);
		
hipEventRecord(start, NULL);
		hipMemcpy(gpu_frame, frame, FRAME_ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice); 
		hipMemcpy(gpu_background, background, FRAME_ARRAY_SIZE * sizeof(double), hipMemcpyHostToDevice); 
		correct3<<<grid_size, block_size>>>(gpu_frame, gpu_background,gpu_profile);
		hipMemcpy(frame, gpu_frame, FRAME_ARRAY_SIZE * sizeof(int), hipMemcpyDeviceToHost); 
			hipEventRecord(stop, NULL);
			hipEventSynchronize(stop);
			 elapsedTime=-1;
			hipEventElapsedTime(&elapsedTime, start, stop);
			printf("Run time is: %f \n",elapsedTime);
hipEventRecord(start, NULL);
		hipMemcpy(gpu_frame, frame, FRAME_ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice); 
		hipMemcpy(gpu_background, background, FRAME_ARRAY_SIZE * sizeof(double), hipMemcpyHostToDevice); 
		correct3<<<grid_size, block_size>>>(gpu_frame, gpu_background,gpu_profile);
		hipMemcpy(frame, gpu_frame, FRAME_ARRAY_SIZE * sizeof(int), hipMemcpyDeviceToHost); 
			hipEventRecord(stop, NULL);
			hipEventSynchronize(stop);
			 elapsedTime=-1;
			hipEventElapsedTime(&elapsedTime, start, stop);
			printf("Run time is: %f \n",elapsedTime);
hipEventRecord(start, NULL);
		hipMemcpy(gpu_frame, frame, FRAME_ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice); 
		hipMemcpy(gpu_background, background, FRAME_ARRAY_SIZE * sizeof(double), hipMemcpyHostToDevice); 
		correct3<<<grid_size, block_size>>>(gpu_frame, gpu_background,gpu_profile);
		hipMemcpy(frame, gpu_frame, FRAME_ARRAY_SIZE * sizeof(int), hipMemcpyDeviceToHost); 
			hipEventRecord(stop, NULL);
			hipEventSynchronize(stop);
			 elapsedTime=-1;
			hipEventElapsedTime(&elapsedTime, start, stop);
			printf("Run time is: %f \n",elapsedTime);
hipEventRecord(start, NULL);
		hipMemcpy(gpu_frame, frame, FRAME_ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice); 
		hipMemcpy(gpu_background, background, FRAME_ARRAY_SIZE * sizeof(double), hipMemcpyHostToDevice); 
		correct3<<<grid_size, block_size>>>(gpu_frame, gpu_background,gpu_profile);
		hipMemcpy(frame, gpu_frame, FRAME_ARRAY_SIZE * sizeof(int), hipMemcpyDeviceToHost); 
			hipEventRecord(stop, NULL);
			hipEventSynchronize(stop);
			 elapsedTime=-1;
			hipEventElapsedTime(&elapsedTime, start, stop);
			printf("Run time is: %f \n",elapsedTime);
hipEventRecord(start, NULL);
		hipMemcpy(gpu_frame, frame, FRAME_ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice); 
		hipMemcpy(gpu_background, background, FRAME_ARRAY_SIZE * sizeof(double), hipMemcpyHostToDevice); 
		correct3<<<grid_size, block_size>>>(gpu_frame, gpu_background,gpu_profile);
		hipMemcpy(frame, gpu_frame, FRAME_ARRAY_SIZE * sizeof(int), hipMemcpyDeviceToHost); 
			hipEventRecord(stop, NULL);
			hipEventSynchronize(stop);
			 elapsedTime=-1;
			hipEventElapsedTime(&elapsedTime, start, stop);
			printf("Run time is: %f \n",elapsedTime);
hipEventRecord(start, NULL);
		hipMemcpy(gpu_frame, frame, FRAME_ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice); 
		hipMemcpy(gpu_background, background, FRAME_ARRAY_SIZE * sizeof(double), hipMemcpyHostToDevice); 
		correct3<<<grid_size, block_size>>>(gpu_frame, gpu_background,gpu_profile);
		hipMemcpy(frame, gpu_frame, FRAME_ARRAY_SIZE * sizeof(int), hipMemcpyDeviceToHost); 
			hipEventRecord(stop, NULL);
			hipEventSynchronize(stop);
			 elapsedTime=-1;
			hipEventElapsedTime(&elapsedTime, start, stop);
			printf("Run time is: %f \n",elapsedTime);
hipEventRecord(start, NULL);
		hipMemcpy(gpu_frame, frame, FRAME_ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice); 
		hipMemcpy(gpu_background, background, FRAME_ARRAY_SIZE * sizeof(double), hipMemcpyHostToDevice); 
		correct3<<<grid_size, block_size>>>(gpu_frame, gpu_background,gpu_profile);
		hipMemcpy(frame, gpu_frame, FRAME_ARRAY_SIZE * sizeof(int), hipMemcpyDeviceToHost); 
			hipEventRecord(stop, NULL);
			hipEventSynchronize(stop);
			 elapsedTime=-1;
			hipEventElapsedTime(&elapsedTime, start, stop);
			printf("Run time is: %f \n",elapsedTime);
hipEventRecord(start, NULL);
		hipMemcpy(gpu_frame, frame, FRAME_ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice); 
		hipMemcpy(gpu_background, background, FRAME_ARRAY_SIZE * sizeof(double), hipMemcpyHostToDevice); 
		correct3<<<grid_size, block_size>>>(gpu_frame, gpu_background,gpu_profile);
		hipMemcpy(frame, gpu_frame, FRAME_ARRAY_SIZE * sizeof(int), hipMemcpyDeviceToHost); 
			hipEventRecord(stop, NULL);
			hipEventSynchronize(stop);
			 elapsedTime=-1;
			hipEventElapsedTime(&elapsedTime, start, stop);
			printf("Run time is: %f \n",elapsedTime);
hipEventRecord(start, NULL);
		hipMemcpy(gpu_frame, frame, FRAME_ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice); 
		hipMemcpy(gpu_background, background, FRAME_ARRAY_SIZE * sizeof(double), hipMemcpyHostToDevice); 
		correct3<<<grid_size, block_size>>>(gpu_frame, gpu_background,gpu_profile);
		hipMemcpy(frame, gpu_frame, FRAME_ARRAY_SIZE * sizeof(int), hipMemcpyDeviceToHost); 
			
		
		
	
		
		// Stop event
		hipEventRecord(stop, NULL);
		hipEventSynchronize(stop);

		elapsedTime=-1;
		hipEventElapsedTime(&elapsedTime, start, stop);
		
		printf("Run time is: %f \n",elapsedTime);
		
				
		//5- copy the corrected image back to the CPU
		hipMemcpy(frame, gpu_frame, FRAME_ARRAY_SIZE * sizeof(int), hipMemcpyDeviceToHost); 
		
	
				
	}
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	for(int index = 0 ; index < 3 ; index++)
	printf("%d\n", frame[index]);
	
	end = clock();
	runTime = (end-tstart);
	printf("total Run time is %g seconds\n",runTime);
	hipFree( gpu_profile );
	hipFree( gpu_frame );
	hipFree( gpu_background );

	free(profile);
	free(frame );
	free(background );

	//system("PAUSE");	
	// stop outer timer
	// print outer timer -- 3333 millisenconds MAX
	return EXIT_SUCCESS;
}

int main1(int argc, char** argv)
{

	return 0;
}