#include <stdio.h>
#include <time.h>
#include <math.h>
#include <float.h>

#include "hip/hip_runtime.h"


 
const int blocksize = 1; 
const int N = 16; 


const int PROFILE_SIZE = 8376;
const int PROFILE_ARRAY_SIZE = PROFILE_SIZE * 6;
__device__ const int GPU_PROFILE_SIZE = 8376;

const int FRAME_DIMENSIONS = 800 * 600;
const int FRAME_ARRAY_SIZE = FRAME_DIMENSIONS * 3;
__device__ const int GPU_FRAME_DIMENSIONS = 800 * 600;

__device__ double X,Y,Z,L,A,B;

__device__ double distance(double L1,double A1,double B1,double L2,double A2,double B2)
{
	 double l, a, b, result,sqresult;
      l = L1 - L2;
      a = A1 - A2;
      b = B1 - B2;
      l = l * l;
      a = a * a;
      b = b * b;
	  result = l + a + b;

	  sqresult = sqrt(result);
	  return sqresult;
}

__device__ void addXYZ(double X1,double Y1,double Z1,double X2,double Y2,double Z2)
{
	X=X1+X2;
	Y=Y1+Y2;
	Z=Z1+Z2;
/*
	if(X>0.9504)
		X=0.9504;
	else if (X<0)
		X=0;
	else
		X=X;
	
	if(X>1)
		Y=1
	else if (Y<0)
		Y=0;
	else
		Y=Y;

	if(Z>1.0888)
		Z=1.0888;
	else if (Z<0)
		Z=0;
	else
		Z=Z;
*/
}

// rgb to xyz
//__device__ void RGBToXYZ(unsigned char R,unsigned char G,unsigned char B )
__device__ void RGBToXYZ(int R,int G,int B )
    {
      // by the formula given the the web page http://www.brucelindbloom.com/index.html [XYZ]=[M][RGB]
      //In order to properly use this matrix, the RGB values must be linear and in the nominal range [0.0, 1.0].
      // RGB values may first need conversion (for example, dividing by 255 and then raising them to a power)
      // Where M for D65:	 0.4124564  0.3575761  0.1804375
      //0.2126729  0.7151522  0.0721750
      //0.0193339  0.1191920  0.9503041

      //// to make rgb values linear red, green, blue values
      double rLinear = (double)R / 255.0;
      double gLinear = (double)G / 255.0;
      double bLinear = (double)B / 255.0;

      // convert to a sRGB form

      //double r =  Math.pow((rLinear ), 2.2) ;
      //double g =  Math.pow((gLinear ), 2.2) ;
      //double b = Math.pow((bLinear ), 2.2) ;
      double r, g, b;

      if (rLinear > 0.04045)
        r = pow(((rLinear + 0.055) / 1.055), 2.2);
      else
        r = rLinear / 12.92;

      if (gLinear > 0.04045)
        g = pow(((gLinear + 0.055) / 1.055), 2.2);
      else
        g = gLinear / 12.92;

      if (bLinear > 0.04045)
        b = pow(((bLinear + 0.055) / 1.055), 2.2);
      else
        b = bLinear / 12.92;


		X=(r * 0.4124564 + g * 0.3575761 + b * 0.1804375);
        Y=(r * 0.2126729 + g * 0.7151522 + b * 0.0721750);
        Z=(r * 0.0193339 + g * 0.1191920 + b * 0.9503041);

		/*
	if(X>0.9504)
		X=0.9504;
	else if (X<0)
		X=0;
	else
		X=X;
	
	if(X>1)
		Y=1
	else if (Y<0)
		Y=0;
	else
		Y=Y;

	if(Z>1.0888)
		Z=1.0888;
	else if (Z<0)
		Z=0;
	else
		Z=Z;
*/
    }

__device__  double FX(double e)
    {
		if(e > 0.008856) 
			e=pow(e, (1.0 / 3.0));
		else
			e=(903.3 * e + 16) / 116;
			return e;
    }

__device__ double Lxyz(double e)
    {
      if (e > 0.008856) 
	  e= (116 * pow(e, (1.0 / 3.0))) - 16 ;
	  else
	  e= (903.3 * e);
	  return e;
    }

//__device__ void ToLAB(unsigned char R,unsigned char G,unsigned char B )
__device__ void ToLAB(int R,int G,int B )
    {
      double Fx, Fy, Fz;
     
      RGBToXYZ(R,G,B);

      double yr = Y / 1.0000;
      double xr = X /0.9504;
      double zr = Z / 1.0888;

      Fx = FX(xr);
      Fy = FX(yr);
      Fz = FX(zr);

      L = Lxyz(yr);
      A = 500 * (Fx - Fy);
      B = 200 * (Fy - Fz);

     
    }

__device__ void XYZtoLAB(double X1, double Y1,double Z1)
{
	  double Fx, Fy, Fz;
           
      double yr = Y1 / 1.0000;
      double xr = X1 /0.9504;
      double zr = Z1 / 1.0888;

      Fx = FX(xr);
      Fy = FX(yr);
      Fz = FX(zr);

      L = Lxyz(yr);
      A = 500 * (Fx - Fy);
      B = 200 * (Fy - Fz);
}

__device__ void LABToXYZ(double L1,double A1,double B1)
    {
      double  xr, yr, zr, Xr, Yr, Zr, Fx, Fy, Fz;

      Fy = (L1 + 16) / 116;

      Fx = (A1 / 500) + Fy;

      Fz = Fy - (B1 / 200);

      if (pow(Fx, 3) > (0.008856)) 
		xr=pow(Fx, 3);
	  else
	    xr=((116 * Fx - 16) / 903.3);

      if(L1 > (0.008856 * 903.3))
		yr= pow(Fy, 3) ; 
	  else
		yr=L1 / 903.3;

      if (pow(Fz, 3) > (0.008856)) 
		zr= pow(Fz, 3);
	  else
		zr= ((116 * Fz - 16) / 903.3);

      //for D65
      Xr = 0.9504;

      Yr = 1;

      Zr = 1.0888;

      X = xr * Xr;
      Y = yr * Yr;
      Z = zr * Zr;

	/*
	if(X>0.9504)
		X=0.9504;
	else if (X<0)
		X=0;
	else
		X=X;
	
	if(X>1)
		Y=1
	else if (Y<0)
		Y=0;
	else
		Y=Y;

	if(Z>1.0888)
		Z=1.0888;
	else if (Z<0)
		Z=0;
	else
		Z=Z;
*/
	  }
	
 __global__ void correct(int *block_frame, double *block_background, double *block_profile,char *a, int *b) 
{
	a[threadIdx.x] += b[threadIdx.x];
	//GPU_FRAME_DIMENSIONS
	for(int pixel = 0 ; pixel < 1 ; pixel++)
	{
		int R = block_frame[3*pixel + 0];
		int G = block_frame[3*pixel + 1];
		int B = block_frame[3*pixel + 2];

		double bgX = block_background[3*pixel + 0];
		double bgY = block_background[3*pixel + 1];
		double bgZ = block_background[3*pixel + 2];

		/*double bgX1 = block_background[3*pixel + 0];
		double bgY1 = block_background[3*pixel + 1];
		double bgZ1 = block_background[3*pixel + 2];

		block_frame[3*pixel + 0]=(int)(block_background[3*pixel + 0]*block_background[3*pixel + 0]);
		block_frame[3*pixel + 1]=(int)(block_background[3*pixel + 1]*block_background[3*pixel + 1]);
		block_frame[3*pixel + 2]=(int)(block_background[3*pixel + 2]*block_background[3*pixel + 2]);

	}*/

		
		double FrameX,FrameY,FrameZ,FrameL,FrameA,FrameB,BlendL,BlendA,BlendB;
		
		
		// to find the best fit bin color
		// SAVES LAB in device global L,A,B variables
		ToLAB(R,G,B);
		FrameL=L;
		FrameA=A;
		FrameB=B;

		double DistanceInBin =  DBL_MIN;
		double ClosestBinDistance;
		int BinIndex=0;
		
		double keyL;
		double keyA;
		double keyB;

		for(int bin = 0 ; bin < PROFILE_SIZE ; bin++)
		{			
			 keyL = block_profile[6*bin + 0];
			 keyA = block_profile[6*bin + 1];
			 keyB = block_profile[6*bin + 2];

			 ClosestBinDistance=distance(FrameL,FrameA,FrameB,keyL,keyA,keyB);

			 if (DistanceInBin >= ClosestBinDistance)
	               continue;
				
	          DistanceInBin = ClosestBinDistance;
			  BinIndex=bin;
		}

		FrameL=block_profile[6*BinIndex + 3];
		FrameA=block_profile[6*BinIndex + 4];
		FrameB=block_profile[6*BinIndex + 5];

		//DO YOUR MAGIC
		DistanceInBin =  DBL_MIN;
		ClosestBinDistance=0;
		BinIndex=0;
		for(int bin = 0 ; bin < PROFILE_SIZE ; bin++)
		{			
			
			double valueL = block_profile[6*bin + 3];
			double valueA = block_profile[6*bin + 4];
			double valueB = block_profile[6*bin + 5];

			//getting the xyz values of the chocen bin
			LABToXYZ(valueL,valueA,valueB);

			FrameX=X;
			FrameY=Y;
			FrameZ=Z;

			addXYZ(FrameX,FrameY,FrameZ,bgX,bgY,bgZ);
			
			XYZtoLAB(X,Y,Z);
			BlendL=L;
			BlendA=A;
			BlendB=B;


			ClosestBinDistance=distance(FrameL,FrameA,FrameB,BlendL,BlendA,BlendB);

			 if (DistanceInBin >= ClosestBinDistance)
	               continue;
				
	             DistanceInBin = ClosestBinDistance;
			  BinIndex=bin;

		}

		block_frame[3*pixel + 0]=(int)block_profile[6*BinIndex + 3];
		block_frame[3*pixel + 1]=(int)block_profile[6*BinIndex + 4];
		block_frame[3*pixel + 2]=(int)block_profile[6*BinIndex + 5];
	
	}
}

//basic cuda whihc runs on a single thread 
int main(int argc, char** argv)
{
	//
	char a[N] = "Hello \0\0\0\0\0\0";
	int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

	char *ad;
	int *bd;
	const int csize = N*sizeof(char);
	const int isize = N*sizeof(int);
 
	printf("%s", a);

	///0
	clock_t tstart;
	clock_t end;
	double runTime;

	double *background,*profile;
	//unsigned char *frame;
	int *frame;

	//var display-profile -- lookup table in LAB
	profile = (double*) malloc(PROFILE_ARRAY_SIZE * sizeof(double));

	for(int index = 0 ; index < PROFILE_ARRAY_SIZE ; index++)
		profile[index] = 30;

	//var frame-image -- image to correct in RGB
	/*frame=	(unsigned char*)malloc(FRAME_ARRAY_SIZE * sizeof(unsigned char));
	for(int index = 0 ; index < FRAME_ARRAY_SIZE ; index++)
		frame[index] = '3';
	*/

	frame =	(int*)malloc(FRAME_ARRAY_SIZE * sizeof(int));
	for(int index = 0 ; index < FRAME_ARRAY_SIZE ; index++)
		frame[index] = 1;
	//var background-image -- background image in YXZ
	background= (double*)malloc(FRAME_ARRAY_SIZE * sizeof(double));
	for(int index = 0 ; index < FRAME_ARRAY_SIZE ; index++)
		background[index] = 0;

	//pointers on the device
	double *gpu_profile;
	//unsigned char *gpu_frame;
	int *gpu_frame;

	double *gpu_background;
	printf("prg starting\n");

	const int psize = PROFILE_ARRAY_SIZE * sizeof(double);
	//const int fsize = FRAME_ARRAY_SIZE * sizeof(unsigned char);
	const int fsize = FRAME_ARRAY_SIZE * sizeof(int);
	const int bgsize = FRAME_ARRAY_SIZE * sizeof(double);
	
	//memory allocation on the GPU
	hipMalloc(&gpu_profile, PROFILE_ARRAY_SIZE * sizeof(double)); 
	//cudaMalloc(&gpu_frame, FRAME_ARRAY_SIZE * sizeof(unsigned char)); 
	hipMalloc(&gpu_frame, FRAME_ARRAY_SIZE * sizeof(int)); 
	hipMalloc(&gpu_background, FRAME_ARRAY_SIZE * sizeof(double)); 

	//1- copy the profile
	hipMemcpy(gpu_profile, profile, psize, hipMemcpyHostToDevice); 

	// start outer timer

	//-- for 100 frames
	for(int f = 0 ; f < 1 ; f++)
	{
		//start inner timer
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		
		//2- pass the image to correct to the GPU
		//cudaMemcpy(gpu_frame, frame, FRAME_ARRAY_SIZE * sizeof(unsigned char), cudaMemcpyHostToDevice); 
		hipMemcpy(gpu_frame, frame, FRAME_ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice); 

		//3- pass the background image to the GPU
		hipMemcpy(gpu_background, background, FRAME_ARRAY_SIZE * sizeof(double), hipMemcpyHostToDevice); 
		
		////////////////////
		hipMalloc((void**)&ad, csize ); 
		hipMalloc((void**)&bd, isize ); 
		hipMemcpy(ad, a, csize, hipMemcpyHostToDevice ); 
		hipMemcpy(bd, b, isize, hipMemcpyHostToDevice ); 
		/////////////////////
		//4- call the kernel
		dim3 dimBlock( blocksize, 1 );
		dim3 dimGrid( 1, 1 );
	
		tstart = clock();
				// Start record
		hipEventRecord(start, 0);
		correct<<<dimGrid, dimBlock>>>(gpu_frame, gpu_background,gpu_profile,ad, bd);
		// Stop event
		hipEventRecord(stop, 0);
		
		hipEventSynchronize(stop);

		float elapsedTime=-1;
		hipEventElapsedTime(&elapsedTime, start, stop);
		// that's our time!
		
		printf("Run time is: %f \n",elapsedTime);
		// Clean up:
		hipEventDestroy(start);
		hipEventDestroy(stop);
		
		end = clock();
	
		runTime = ((end-tstart));
		///////////////////////////
		hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost ); 
		hipFree(ad);
		hipFree(bd);
		
		printf("%s\n", a);
		/////////////////////////////////
		//cudaMemcpy(frame, gpu_frame, FRAME_ARRAY_SIZE * sizeof(unsigned char), cudaMemcpyDeviceToHost); 
		hipMemcpy(frame, gpu_frame, FRAME_ARRAY_SIZE * sizeof(int), hipMemcpyDeviceToHost); 
		//cudaMemcpy(background, gpu_background, FRAME_ARRAY_SIZE * sizeof(unsigned char), cudaMemcpyDeviceToHost); 
		hipFree( gpu_frame );
		hipFree( gpu_background );

		//5- copy the corrected image back to the CPU
		
		for(int index = 0 ; index < 3 ; index++)
		printf("%d\n", frame[index]);
		//stop inner timer
		//print inner timer -- 33 milliseconds MAX
	}

	//printf("Run time is %g seconds\n",runTime);
	hipFree( gpu_profile );
	//system("PAUSE");	
	// stop outer timer
	// print outer timer -- 3333 millisenconds MAX
	return EXIT_SUCCESS;
}

int main1(int argc, char** argv)
{
	return 0;
}